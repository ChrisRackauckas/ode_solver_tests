#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <string>
#include <sstream>
#include <fstream>

#include "KellerMiksis_1f.cuh"
#include "SingleSystem_PerThread.cuh"

#define PI 3.14159265358979323846

using namespace std;

// Physical control parameters
const int NumberOfFrequency = 128;
	
// Solver Configuration
#define SOLVER RKCK45 // RK4, RKCK45
const int NT   = NumberOfFrequency; // NumberOfThreads
const int SD   = 2;     // SystemDimension
const int NCP  = 9;    // NumberOfControlParameters
const int NSP  = 0;     // NumberOfSharedParameters
const int NISP = 0;     // NumberOfIntegerSharedParameters
const int NE   = 0;     // NumberOfEvents
const int NA   = 1;     // NumberOfAccessories
const int NIA  = 0;     // NumberOfIntegerAccessories
const int NDO  = 0;     // NumberOfPointsOfDenseOutput

void Linspace(vector<double>&, double, double, int);
void Logspace(vector<double>&, double, double, int);
void FillSolverObject(ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,double>&, const vector<double>&, int);

int main()
{
	int BlockSize = 64;
	
	vector<double> Frequency(NT,0);
	Logspace(Frequency, 20.0, 1000.0, NT);
	
	// Setup CUDA a device
	ListCUDADevices();
	
	int MajorRevision  = 3;
	int MinorRevision  = 5;
	int SelectedDevice = SelectDeviceByClosestRevision(MajorRevision, MinorRevision);
	
	PrintPropertiesOfSpecificDevice(SelectedDevice);
	
	// Setup Solver
	ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,double> ScanKellerMiksis(SelectedDevice);
	
	ScanKellerMiksis.SolverOption(ThreadsPerBlock, BlockSize);
	ScanKellerMiksis.SolverOption(RelativeTolerance, 0, 1e-10);
	ScanKellerMiksis.SolverOption(RelativeTolerance, 1, 1e-10);
	ScanKellerMiksis.SolverOption(AbsoluteTolerance, 0, 1e-10);
	ScanKellerMiksis.SolverOption(AbsoluteTolerance, 1, 1e-10);
	
// SIMULATIONS ------------------------------------------------------------------------------------
	
	vector<double> GlobalMaxima(NT,0);
	FillSolverObject(ScanKellerMiksis, Frequency, NT);
	
	clock_t SimulationStart = clock();
	ScanKellerMiksis.SynchroniseFromHostToDevice(All);
	for (int i=0; i<1024; i++)
	{
		ScanKellerMiksis.Solve();
		ScanKellerMiksis.InsertSynchronisationPoint();
		ScanKellerMiksis.SynchroniseSolver();
	}
	ScanKellerMiksis.SynchroniseFromDeviceToHost(All);
	ScanKellerMiksis.InsertSynchronisationPoint();
	ScanKellerMiksis.SynchroniseSolver();
	clock_t TransientSimulationEnd = clock();
		cout << "Transient simulation time: " << (double)(TransientSimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "s" << endl << endl;
	
	for (int i=0; i<64; i++)
	{
		ScanKellerMiksis.Solve();
		ScanKellerMiksis.SynchroniseFromDeviceToHost(Accessories);
		ScanKellerMiksis.InsertSynchronisationPoint();
		ScanKellerMiksis.SynchroniseSolver();
		
		for (int tid=0; tid<NT; tid++)
			GlobalMaxima[tid] = fmax( ScanKellerMiksis.GetHost(tid, Accessories, 0), GlobalMaxima[tid] );
	}
	
	// Save collected data to file
	ofstream DataFile;
	DataFile.open ( "KellerMiksis_1f.txt" );
	int Width = 18;
	DataFile.precision(10);
	DataFile.flags(ios::scientific);
	
	for (int tid=0; tid<NT; tid++)
	{
		DataFile.width(8); DataFile << tid << ',';
		DataFile.width(Width); DataFile << Frequency[tid] << ',';
		DataFile.width(Width); DataFile << ScanKellerMiksis.GetHost(tid, ActualState, 0) << ',';
		DataFile.width(Width); DataFile << ScanKellerMiksis.GetHost(tid, ActualState, 1) << ',';
		DataFile.width(Width); DataFile << GlobalMaxima[tid];
		DataFile << '\n';
	}
	
	DataFile.close();
	clock_t TotalSimulationEnd = clock();
		cout << "Total simulation time: " << (double)(TotalSimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "s" << endl << endl;
	
	//ScanKellerMiksis.Print(DenseOutput, 25);
}

// ------------------------------------------------------------------------------------------------

void Linspace(vector<double>& x, double B, double E, int N)
{
    double Increment;
	
	x[0] = B;
	
	if ( N>1 )
	{
		x[N-1] = E;
		Increment = (E-B)/(N-1);
		
		for (int i=1; i<N-1; i++)
		{
			x[i] = B + i*Increment;
		}
	}
}

void Logspace(vector<double>& x, double B, double E, int N)
{
    x[0] = B; 
	
	if ( N>1 )
	{
		x[N-1] = E;
		double ExpB = log10(B);
		double ExpE = log10(E);
		double ExpIncr = (ExpE-ExpB)/(N-1);
		for (int i=1; i<N-1; i++)
		{
			x[i] = pow(10,ExpB + i*ExpIncr);
		}
	}
}

// ------------------------------------------------------------------------------------------------

void FillSolverObject(ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,double>& Solver, const vector<double>& Values, int NumberOfThreads)
{
	// Declaration of physical control parameters
	double P2; // frequency          [kHz]
	
	// Declaration of constant parameters
	double P1 =  1.5; // pressure amplitude   [bar]
	double P6 = 10.0; // equilibrium radius   [mum]
	double P7 =  1.0; // ambient pressure     [bar]
	double P9 =  1.4; // polytrophic exponent [-]
	
	// Material properties
	double Pv  = 3.166775638952003e+03;
    double Rho = 9.970639504998557e+02;
    double ST  = 0.071977583160056;
    double Vis = 8.902125058209557e-04;
    double CL  = 1.497251785455527e+03;
	
	// Auxiliary variables
	double Pinf;
	double PA1;
	double RE;
	double f1;
	
	for (int i=0; i<NumberOfThreads; i++)
	{	
		// Update physical parameters
		P2 = Values[i]; // frequency [kHz]
		
		Solver.SetHost(i, TimeDomain, 0, 0);
		Solver.SetHost(i, TimeDomain, 1, 1);
		
		// Initial conditions are the equilibrium condition y1=1; y2=0;
		Solver.SetHost(i, ActualState, 0, 1.0);
		Solver.SetHost(i, ActualState, 1, 0.0);
		
		// Scaling of physical parameters to SI
		Pinf = P7 * 1e5;
		PA1  = P1 * 1e5;
		RE   = P6 / 1e6;
		
		// Scale to angular frequency
		f1   = 2.0*PI*(P2*1000);
					
		// System coefficients and other, auxiliary parameters
		Solver.SetHost(i, ControlParameters, 0, (2.0*ST/RE + Pinf - Pv) * pow(2.0*PI/RE/f1, 2.0) / Rho );
		Solver.SetHost(i, ControlParameters, 1, (1.0-3.0*P9) * (2*ST/RE + Pinf - Pv) * (2.0*PI/RE/f1) / CL/Rho );
		Solver.SetHost(i, ControlParameters, 2, (Pinf - Pv) * pow(2.0*PI/RE/f1, 2.0) / Rho );
		Solver.SetHost(i, ControlParameters, 3, (2.0*ST/RE/Rho) * pow(2.0*PI/RE/f1, 2.0) );
		Solver.SetHost(i, ControlParameters, 4, (4.0*Vis/Rho/pow(RE,2.0)) * (2.0*PI/f1) );
		Solver.SetHost(i, ControlParameters, 5, PA1 * pow(2.0*PI/RE/f1, 2.0) / Rho );
		Solver.SetHost(i, ControlParameters, 6, (RE*f1*PA1/Rho/CL) * pow(2.0*PI/RE/f1, 2.0) );
		Solver.SetHost(i, ControlParameters, 7, RE*f1/(2.0*PI)/CL );
		Solver.SetHost(i, ControlParameters, 8, 3.0*P9 );
	}
}