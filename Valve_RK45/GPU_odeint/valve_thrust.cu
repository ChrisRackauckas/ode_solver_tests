#include "hip/hip_runtime.h"
//============================================================================
// Name        : valve_thrust.cu
// Author      : Lambert Plavecz
// Version     :
// Copyright   : no
// Description : Simulation of a pressure relief valve with odeint CUDA Thrust RKCK54
//============================================================================


#include <iostream>
#include <fstream>
#include <iomanip>
#include <string>
#include <chrono>
#include <cmath>
#include <utility>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>


using namespace std;
using namespace boost::numeric::odeint;

typedef double value_type;
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< int > int_vector;

int num = 12; //12 - 5952528 ms
value_type inv_num = 1.0/(num-1);

string file_name = "impact_dyn_output_thrust.txt";

const value_type kappa = 1.25;
const value_type beta = 20.0;
const value_type delta = 10.0;
const value_type r = 0.8;


struct impact_dynamics
{
    struct impact_functor
    {
        template< class T >
        __host__ __device__
        void operator()( T t )const
        {												
            value_type q = thrust::get< 3 >( t );
            value_type x = thrust::get< 0 >( t );
            value_type y = thrust::get< 1 >( t );
            value_type z = thrust::get< 2 >( t );
            thrust::get< 4 >( t ) = y;
            thrust::get< 5 >( t ) = -kappa * y - (x + delta) + z;
            thrust::get< 6 >( t ) = beta * (q - x * sqrt(z));

        }
    };

    impact_dynamics( size_t N , const state_type &q)
    : m_N( N ) , m_q( q ) { }

    template< class State , class Deriv >
    void operator()(  const State &x , Deriv &dxdt , value_type t ) const
    {
        thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple(
                        x.begin() ,
                        x.begin() + m_N ,
                        x.begin() + 2 * m_N ,
                        m_q.begin() ,
                        dxdt.begin() ,
                        dxdt.begin() + m_N ,
                        dxdt.begin() + 2 * m_N  ) ) ,
                thrust::make_zip_iterator( thrust::make_tuple(
                        x.begin() + m_N ,
                        x.begin() + 2 * m_N ,
                        x.end() ,
                        m_q.end() ,
                        dxdt.begin() + m_N ,
                        dxdt.begin() + 2 * m_N ,
                        dxdt.end()  ) ) ,
                impact_functor() );
    }
    size_t m_N;
    const state_type &m_q;
};

class impact_observer
{
public:

	struct observer_functor
    {
		
		template< class T >
        __host__ __device__
        void operator()( T t )
        {
			int count = thrust::get< 3 >( t );
			if(count > 2048+63)return;
			value_type extr_prev = thrust::get< 4 >( t );
            value_type x = thrust::get< 0 >( t );
            value_type y = thrust::get< 1 >( t );
			value_type y_prev = thrust::get< 2 >( t );
			int extr_place = thrust::get< 5 >( t );
			if(x <= 0 && y < 0){
				y = -r * y;
				thrust::get< 1 >( t ) = y;
			}
			if(y*y_prev < 0){//extremum
				if(2047 < count){
					thrust::get< 6 >( t ) = x;
					thrust::get< 5 >( t ) = extr_place+1;
				}
				count++;
				if(fabs(extr_prev - x) < 1.0e-6 && fabs(x) > 1.0e-6 && y_prev!=0.0){ //convergence detection
					thrust::get< 6 >( t ) = x;
					count = 5000;
				}
				thrust::get< 4 >( t ) = x;
				thrust::get< 3 >( t ) = count;
			}
			thrust::get< 2 >( t ) = y;

        }
		observer_functor(){}
    };
	
    impact_observer(size_t N, state_type &extrema, int_vector &extr_places)
		: m_N( N ), m_extr_places(extr_places), m_extrema(extrema)
	{
		y_prev = state_type(N);
		thrust::fill(y_prev.begin(), y_prev.end(), 0.0);
		m_count = int_vector(N);
		thrust::fill(m_count.begin(), m_count.end(), 0);
		extr_prev = state_type(N);
		thrust::fill(extr_prev.begin(), extr_prev.end(), 10000.0); //arbitrary large number
    }

	template< class State >
    void operator()( State &x, double t )
	//(const state_type &x , value_type t )
	{
		thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple(
                        x.begin() ,
                        x.begin() + m_N ,
						y_prev.begin(),
						m_count.begin() ,
						extr_prev.begin(),
						m_extr_places.begin(),
						thrust::make_permutation_iterator(m_extrema.begin(),m_extr_places.begin())   ) ),
                thrust::make_zip_iterator( thrust::make_tuple(
                        x.begin() + m_N ,
                        x.begin() + 2 * m_N ,
						y_prev.end() ,
						m_count.end() ,
						extr_prev.end(),
						m_extr_places.end(),
						thrust::make_permutation_iterator(m_extrema.begin(),m_extr_places.end())  ) ) ,
                obs_func );
		
		auto min_pos = thrust::min_element(m_count.begin(),m_count.end())-m_count.begin();
		if(min_count < m_count[min_pos]){//cout << m_count[min_pos] << endl;
			min_count = m_count[min_pos];
			if(min_count%200==0) cout << min_count << endl;
			//if( min_count == 1366) for(int z=0;z < m_N; z++) cout << m_count[z] << endl;
		}
		if(m_count[min_pos] > 2048+63) throw 1;
    }
private:	
	state_type y_prev;
	state_type extr_prev;
	state_type &m_extrema; //pointer to device memory
	size_t m_N;
	int min_count = 0;
	int_vector m_count;
	int_vector &m_extr_places;
	observer_functor obs_func;
};
int nums[18] = {256, 768, 1536, 3072, 3840, 5120, 7680, 15360, 30720, 46080, 61440, 76800, 92160, 122880, 184320, 307200, 768000, 4147200};
//343767 370154 389796 410224 398229 250804 453742 0 700243

int main() {
	cout << "Impact dynamics started" << endl;

	//TODO try dopri5
	typedef runge_kutta_cash_karp54< state_type , value_type , state_type , value_type > stepper_type;
	//typedef controlled_runge_kutta< stepper_type, custom_error_checker > controlled_stepper;
	
	/*for(int jj=8; jj < 18; jj++){
		num = nums[jj];
		inv_num = 1.0/num;
		*/
		
	cout << num << endl;
	auto t1 = chrono::high_resolution_clock::now();

	thrust::host_vector< value_type > q_host(num);
	for( int i=0 ; i<num ; i++)
		q_host[i] = 0.2 + value_type(i) * inv_num * 9.8 ;
	
	thrust::host_vector<int> extremum_places_host(num);
	for( int i=0 ; i<num ; i++)
		extremum_places_host[i] = i*64;

	state_type q = q_host;

	state_type x( 3 * num );

	// initialize x
	thrust::fill( x.begin() , x.begin() + num , 0.2 );
	// initialize y,z
	thrust::fill( x.begin() + num, x.end() , 0.0 );
	

	state_type extrema(64 * num);
	thrust::fill( extrema.begin(), extrema.end(), -1.0);
		
	int_vector extremum_places = extremum_places_host;

	impact_dynamics impact( num , q );
	impact_observer obs(num, extrema, extremum_places);
	
	auto stepper = make_controlled( 1.0e-10 , 1.0e-10 , stepper_type());
	
	try{
		integrate_adaptive( stepper , impact, x, 0.0, 1.0e23, 0.01, obs);
	}catch(...){
		//cout << "Enough" << endl;
	}
	thrust::host_vector<value_type> extrema_host(64 * num);
	extrema_host = extrema; //copy back from GPU
	
	ofstream ofs(file_name);
	if(!ofs.is_open())exit(-1);
	ofs.precision(17);
	for(int u = 0;u < num;u++){
		ofs << (0.2 + u * 9.8 * inv_num);
		for(int i=0; i < 64; i++) ofs << " " << extrema_host[u*64+i];
		ofs	<< "\n";
	}

	ofs.flush();
	ofs.close();

	auto t2 = chrono::high_resolution_clock::now();
	cout << "Done" << endl;
	cout << "Time (ms):" << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() << endl;
	
	//}

	return 0;
}
