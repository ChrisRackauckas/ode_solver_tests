#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <string>
#include <fstream>

#include "SingleSystem_PerThread_IndexingMacroEnabled.cuh"
#include "Lorenz_SystemDefinition.cuh"
#include "SingleSystem_PerThread_IndexingMacroDisabled.cuh"
#include "SingleSystem_PerThread.cuh"

using namespace std;

void Linspace(double*, double, double, int);
void Uniform(double*, double, int);

__global__ void RungeKuttaStepOriginal(double* __restrict__, const double* __restrict__, int);
__global__ void RungeKuttaStepRegisterFriendly(double* __restrict__, const double* __restrict__, int);
__device__ void Lorenz(double* __restrict__, const double* __restrict__, double);

int main()
{
// INITIAL SETUP ----------------------------------------------------------------------------------
	
	int NumberOfProblems = 15360*270; // 92160
	int NumberOfThreads  = NumberOfProblems;
	int BlockSize        = 128;
	
	ListCUDADevices();
	
	int MajorRevision  = 3;
	int MinorRevision  = 5;
	int SelectedDevice = SelectDeviceByClosestRevision(MajorRevision, MinorRevision);
	
	PrintPropertiesOfSpecificDevice(SelectedDevice);
	hipSetDevice(SelectedDevice);
	
	double* h_State      = AllocateHostMemory<double>( 3*NumberOfProblems );
	double* h_Parameters = AllocateHostMemory<double>(   NumberOfProblems );
	double* d_State      = AllocateDeviceMemory<double>( 3*NumberOfProblems );
	double* d_Parameters = AllocateDeviceMemory<double>(   NumberOfProblems );
	
	Linspace(h_Parameters, 0.0, 21.0, NumberOfProblems);
	Uniform(h_State, 10.0, NumberOfProblems);
	Uniform(&h_State[   NumberOfProblems ], 10.0, NumberOfProblems);
	Uniform(&h_State[ 2*NumberOfProblems ], 10.0, NumberOfProblems);
	
	
	hipMemcpy(d_State, h_State, 3*sizeof(double)*NumberOfProblems, hipMemcpyHostToDevice);
	hipMemcpy(d_Parameters, h_Parameters, sizeof(double)*NumberOfProblems, hipMemcpyHostToDevice);
	
	
	int GridSize = NumberOfThreads/BlockSize + (NumberOfThreads % BlockSize == 0 ? 0:1);
	
	clock_t SimulationStart;
	clock_t SimulationEnd;
	
	SimulationStart = clock();
		//RungeKuttaStepOriginal<<<GridSize, BlockSize>>> (d_State, d_Parameters, NumberOfProblems);
		RungeKuttaStepRegisterFriendly<<<GridSize, BlockSize>>> (d_State, d_Parameters, NumberOfProblems);
		gpuErrCHK( hipDeviceSynchronize() );
	SimulationEnd = clock();
		cout << "Simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
		cout << "Simulation time / 1000 RK4 step: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC / 10 << "ms" << endl;
		cout << "Ensemble size:                   " << NumberOfProblems << endl << endl;
		
	hipMemcpyAsync(h_State, d_State, 3*sizeof(double)*NumberOfProblems, hipMemcpyDeviceToHost);
	
	
	//for (int i=0; i<NumberOfProblems; i++)
	//	cout << "P: " << h_Parameters[i] << "   Sates: " << h_State[i] << ", " << h_State[i+NumberOfProblems] << ", " << h_State[i+2*NumberOfProblems] << endl;
}

// AUXILIARY FUNCTION -----------------------------------------------------------------------------

void Linspace(double* x, double B, double E, int N)
{
    double Increment;
	
	x[0]   = B;
	
	if ( N>1 )
	{
		x[N-1] = E;
		Increment = (E-B)/(N-1);
		
		for (int i=1; i<N-1; i++)
		{
			x[i] = B + i*Increment;
		}
	}
}

void Uniform(double* x, double V, int N)
{
	for (int i=0; i<N; i++)
	{
		x[i] = V;
	}
}

__forceinline__ __device__ void Lorenz(double* __restrict__ F, const double* __restrict__ X, double P)
{
	// How 5 FMA and 1 ADD/MUL is possible
	F[0] = 10.0*(X[1] - X[0]); // 2 FP inst: 1 FMA, 1 ADD
	F[1] = P*X[0] - X[1] - X[0]*X[2]; // 2 FP inst: 2 FMA
	F[2] = X[0]*X[1] - 2.666 * X[2]; // 2 FP inst: 1 MUL, 1 FMA
}

__global__ void RungeKuttaStepOriginal(double* __restrict__ d_State, const double* __restrict__ d_Parameters, int N)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x; // 1 regs
	
	if (tid < N)
	{
		double X[3]; // 6 regs
		double P;    // 2 regs
		
		double k1[3]; // 6 regs
		double k2[3]; // 6 regs
		double k3[3]; // 6 regs
		double k4[3]; // 6 regs
		double x[3];  // 6 regs
		
		double T    = 0.0;  // 2 regs
		double dT   = 1e-3; // 2 regs
		double dTp2 = 0.5*dT;
		double dTp6 = dT * (1.0/6.0);
		//double t;         // 2 regs
		
		X[0] = d_State[tid];
		X[1] = d_State[tid + N];
		X[2] = d_State[tid + 2*N];
		
		P = d_Parameters[tid];
		
		for (int i=0; i<10000; i++) // 1 regs
		{
			// k1
			Lorenz(k1, X, P); // 5 FMA, 1 ADD/MUL
			
			// k2
			//t = T + 0.5*dT;
			
			#pragma unroll
			for (int j=0; j<3; j++) // 1 regs
				x[j] = X[j] + dTp2*k1[j]; // 3 FMA
			
			Lorenz(k2, x, P); // 5 FMA, 1 ADD/MUL
			
			// k3
			//t = T + 0.5*dT;
			
			#pragma unroll
			for (int j=0; j<3; j++) // 1 regs
				x[j] = X[j] + dTp2*k2[j]; // 3 FMA
			
			Lorenz(k3, x, P); // 5 FMA, 1 ADD/MUL
			
			// k4
			//t = T + dT;
			
			#pragma unroll
			for (int j=0; j<3; j++) // 1 regs
				x[j] = X[j] + dT*k3[j]; // 3 FMA
			
			Lorenz(k4, x, P); // 5 FMA, 1 ADD/MUL
			
			
			// Update state
			#pragma unroll
			for (int j=0; j<3; j++) // 1 regs
				X[j] = X[j] + dTp6*( k1[j] + 2*k2[j] + 2*k3[j] + k4[j] ); // 9 FMA, 3 ADD,
			
			T += dT;
		}
		
		d_State[tid] = X[0];
		d_State[tid + N] = X[1];
		d_State[tid + 2*N] = X[2];
	}
}

__global__ void RungeKuttaStepRegisterFriendly(double* __restrict__ d_State, const double* __restrict__ d_Parameters, int N)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	
	if (tid < N)
	{
		double X[3];
		double P;
		
		double k1[3];
		double ks[3];
		double x[3];
		
		double T    = 0.0;
		double dT   = 1e-3;
		double dTp2 = 0.5*dT;
		double dTp6 = dT * (1.0/6.0);
		//double t;
		
		X[0] = d_State[tid];
		X[1] = d_State[tid + N];
		X[2] = d_State[tid + 2*N];
		
		P = d_Parameters[tid];
		
		for (int i=0; i<10000; i++)
		{
			// k1
			Lorenz(k1, X, P);
			
			// k2
			//t = T + 0.5*dT;
			
			#pragma unroll 3
			for (int j=0; j<3; j++)
			{
				x[j]  = X[j] + dTp2*k1[j];
				ks[j] = k1[j];
			}
			
			Lorenz(k1, x, P);
			
			// k3
			//t = T + 0.5*dT;
			
			#pragma unroll 3
			for (int j=0; j<3; j++)
			{
				x[j]  = X[j] + dTp2*k1[j];
				ks[j] = ks[j]+2*k1[j];
			}
			
			Lorenz(k1, x, P);
			
			// k4
			//t = T + dT;
			
			#pragma unroll 3
			for (int j=0; j<3; j++)
			{
				x[j] = X[j] + dT*k1[j];
				ks[j] = ks[j]+2*k1[j];
			}
			
			Lorenz(k1, x, P);
			
			
			// Update state
			#pragma unroll 3
			for (int j=0; j<3; j++)
				X[j] = X[j] + dTp6*( ks[j] + k1[j] );
			
			T += dT;
		}
		
		d_State[tid] = X[0];
		d_State[tid + N] = X[1];
		d_State[tid + 2*N] = X[2];
	}
}