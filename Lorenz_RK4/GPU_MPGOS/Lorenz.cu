#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <string>
#include <fstream>

#include "SingleSystem_PerThread_IndexingMacroEnabled.cuh"
#include "Lorenz_SystemDefinition.cuh"
#include "SingleSystem_PerThread_IndexingMacroDisabled.cuh"
#include "SingleSystem_PerThread.cuh"

#define PI 3.14159265358979323846

#define SOLVER RK4
#define EVNT   EVNT0
#define DOUT   DOUT0

using namespace std;

void Linspace(vector<double>&, double, double, int);
void FillSolverObject(ProblemSolver<SOLVER,EVNT,DOUT>&, const vector<double>&, int);

int main()
{
// INITIAL SETUP ----------------------------------------------------------------------------------
	
	int NumberOfProblems = 92160;
	int NumberOfThreads  = NumberOfProblems;
	int BlockSize        = 64;
	
	ListCUDADevices();
	
	int MajorRevision  = 3;
	int MinorRevision  = 5;
	int SelectedDevice = SelectDeviceByClosestRevision(MajorRevision, MinorRevision);
	
	PrintPropertiesOfSpecificDevice(SelectedDevice);
	
	int NumberOfParameters_R = NumberOfProblems;
	double R_RangeLower = 0.0;
    double R_RangeUpper = 56.0;
		vector<double> Parameters_R_Values(NumberOfParameters_R,0);
		Linspace(Parameters_R_Values, R_RangeLower, R_RangeUpper, NumberOfParameters_R);
	
	
	ConstructorConfiguration ConfigurationDuffing;
	
	ConfigurationDuffing.NumberOfThreads = NumberOfThreads;
	ConfigurationDuffing.SystemDimension = 3;
	ConfigurationDuffing.NumberOfControlParameters = 1;
	
	ProblemSolver<SOLVER,EVNT,DOUT> ScanDuffing(ConfigurationDuffing, SelectedDevice);
	
	ScanDuffing.SolverOption(ThreadsPerBlock, BlockSize);
	ScanDuffing.SolverOption(InitialTimeStep, 0.001);
	ScanDuffing.SolverOption(MaximumNumberOfTimeSteps, 10000);
	
	
// SIMULATIONS ------------------------------------------------------------------------------------
	
	clock_t SimulationStart;
	clock_t SimulationEnd;
	
	FillSolverObject(ScanDuffing, Parameters_R_Values, NumberOfThreads);
	
	ScanDuffing.SynchroniseFromHostToDevice(All);
	ScanDuffing.InsertSynchronisationPoint();
	ScanDuffing.SynchroniseSolver();
		
	SimulationStart = clock();
		ScanDuffing.Solve();
		ScanDuffing.InsertSynchronisationPoint();
		ScanDuffing.SynchroniseSolver();
	SimulationEnd = clock();
		cout << "Simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
	
	ScanDuffing.SynchroniseFromDeviceToHost(All);
	ScanDuffing.InsertSynchronisationPoint();
	ScanDuffing.SynchroniseSolver();
	
	//for (int i=0; i<NumberOfProblems; i++)
	//	cout << ScanDuffing.GetHost(i, ActualState, 0) << endl;
	
	
	cout << "Test finished!" << endl;
}

// AUXILIARY FUNCTION -----------------------------------------------------------------------------

void Linspace(vector<double>& x, double B, double E, int N)
{
    double Increment;
	
	x[0]   = B;
	
	if ( N>1 )
	{
		x[N-1] = E;
		Increment = (E-B)/(N-1);
		
		for (int i=1; i<N-1; i++)
		{
			x[i] = B + i*Increment;
		}
	}
}

void FillSolverObject(ProblemSolver<SOLVER,EVNT,DOUT>& Solver, const vector<double>& R_Values, int NumberOfThreads)
{
	int ProblemNumber = 0;
	for (int k=0; k<NumberOfThreads; k++)
	{
		Solver.SetHost(ProblemNumber, TimeDomain,  0, 0 );
		Solver.SetHost(ProblemNumber, TimeDomain,  1, 0.001*10000 );
		
		Solver.SetHost(ProblemNumber, ActualState, 0, 10.0 );
		Solver.SetHost(ProblemNumber, ActualState, 1, 10.0 );
		Solver.SetHost(ProblemNumber, ActualState, 2, 10.0 );
		
		Solver.SetHost(ProblemNumber, ControlParameters, 0, R_Values[k] );
		
		ProblemNumber++;
	}
}