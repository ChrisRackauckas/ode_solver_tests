#include "hip/hip_runtime.h"
//============================================================================
// Name        : lorenz_RK4.cpp
// Author      : Lambert Plavecz
// Version     :
// Copyright   : no
// Description : lorenz with odeint CUDA Thrust RK4, Based on the example available at: 
// https://github.com/headmyshoulder/odeint-v2/blob/master/examples/thrust/lorenz_parameters.cu
//============================================================================


#include <iostream>
#include <ctime>
#include <utility>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>


using namespace std;
using namespace boost::numeric::odeint;

typedef double value_type;
typedef thrust::device_vector< value_type > state_type;

int num = 0;
value_type inv_num = 1.0/(num-1);
const int step_number = 1000;

const value_type sigma = 10.0;
const value_type b = 8.0/3.0;
const value_type dt = 0.01;

struct lorenz_system
{
    struct lorenz_functor
    {
        template< class T >
        __host__ __device__
        void operator()( T t )const
        {
            // unpack the parameter we want to vary and the Lorenz variables
            value_type k = thrust::get< 3 >( t );
            value_type x = thrust::get< 0 >( t );
            value_type y = thrust::get< 1 >( t );
            value_type z = thrust::get< 2 >( t );
            thrust::get< 4 >( t ) = sigma * ( y - x );
            thrust::get< 5 >( t ) = k * x - y - x * z;
            thrust::get< 6 >( t ) = -b * z + x * y ;

        }
    };

    lorenz_system( size_t N , const state_type &k )
    : m_N( N ) , m_k( k ) { }

    template< class State , class Deriv >
    void operator()(  const State &x , Deriv &dxdt , value_type t ) const
    {
        thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple(
                        x.begin(),
                        x.begin() + m_N ,
                        x.begin() + 2 * m_N ,
                        m_k.begin() ,
                        dxdt.begin() ,
                        dxdt.begin() + m_N ,
                        dxdt.begin() + 2 * m_N  ) ) ,
                thrust::make_zip_iterator( thrust::make_tuple(
                        x.begin() + m_N ,
                        x.begin() + 2 * m_N ,
                        x.end(),
                        m_k.end() ,
                        dxdt.begin() + m_N ,
                        dxdt.begin() + 2 * m_N ,
                        dxdt.end() ) ) ,
                lorenz_functor() );
    }
    size_t m_N;
    const state_type &m_k;
};

int nums[13] = {256, 512,  1024,  2048,  4096, 7680, 15360, 46080, 92160, 184320, 307200, 768000, 4147200};
int main() {
	cout << "Lorenz RK4 started" << endl;

	runge_kutta4<state_type> stepper;
	
	for(int j=0; j < 13;j++){
		num = nums[j];
		inv_num = 1.0/(num-1);
		cout << num << endl;
		auto t1 = clock();

		vector< value_type > k_host(num);
		for( size_t i=0 ; i<num ; i++)
			k_host[i] = value_type(i) * inv_num * 21.0 ;

		state_type k = k_host;

		state_type x( 3 * num );

		// initialize x,y,z
		thrust::fill( x.begin() , x.end() , 10.0 );

		lorenz_system lorenz( num , k );
	
		integrate_const(stepper, lorenz, x, 0.0, step_number*dt, dt);


		auto t2 = clock();
		cout << "Time (ms):" << (t2 - t1)*1000/CLOCKS_PER_SEC << endl;
	}

	return 0;
}
